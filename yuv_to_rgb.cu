#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <ctime>

using namespace std;

__global__ void RGBToYCbCr(int* r, int* g, int* b) {
    int r1 = float(r);
    int g1 = float(g);
    int b1 = float(b);

    y  = (unsigned char)( 0.299*r1 + 0.587*g1 +  0.114*b1);
    cb = (unsigned char)(-0.169*r1 - 0.331*g1 +  0.499*b1 + 128);
    cr = (unsigned char)( 0.499*r1 - 0.418*g1 - 0.0813*b1 + 128);

    return;
}

__global__ void YCbCrToRGB(int* y, int* cb, int* cr) {
    int r1 = float(y);
    int g1 = float(cb);
    int b1 = float(cr);

    r = y1 + 1.402 * (cr1-128)
    g = y1 - 0.344414*(cb1-128) - 0.71414*(cr1-128)
    b = y1 + 1. 77200*(cb1-128)

    return;
}


int main(int argc, char** argv) {



    return 0;
}

