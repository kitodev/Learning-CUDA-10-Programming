#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <ctime>

using namespace std;

__global__ void RGBToYCbCr(unsigned char r, unsigned char g, unsigned char b) {
    float r1 = (float)(r);
    float g1 = (float)(g);
    float b1 = (float)(b);

    float y  = (float)( 0.299*r1 + 0.587*g1 +  0.114*b1);
    float cb = (float)(-0.169*r1 - 0.331*g1 +  0.499*b1 + 128);
    float cr = (float)( 0.499*r1 - 0.418*g1 - 0.0813*b1 + 128);

    return RGBToYCbCr();
}

__global__ void YCbCrToRGB(float y, float cb, float cr) {
    float r1 = float(y);
    float g1 = float(cb);
    float b1 = float(cr);

    float r = y1 + 1.402 * (b1-128)
    float g = y1 - 0.344414*(g1-128) - 0.71414*(b1-128)
    float b = y1 + 1. 77200*(g1-128)

    return YCbCrToRGB();
}


int main(int argc, char** argv) {



    return 0;
}

